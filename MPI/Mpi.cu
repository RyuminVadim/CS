#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <sstream>
#include <cmath>
#include <ctime>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#include "mpi.h"

#define UPDATE size
#define THREADS_MAX 1024
#define THREAD (size < THREADS_MAX ? size : THREADS_MAX)

int find_area(int size, int gr_size, int DEVICE){
    if (size % gr_size == 0) return size / gr_size;
    
    int tmp = size;
    while (tmp % gr_size != 0){
        tmp++;
    }
    int kek = size - ((gr_size - 1) * tmp / gr_size);
    // std::cout << "DEVICE: "<< DEVICE << "kek: " << kek << std::endl;
    if (kek == 1) {

        if (DEVICE != gr_size - 1) return (tmp/gr_size) -1;
        return gr_size;
    } else if (DEVICE == gr_size - 1) return kek;

    return tmp / gr_size;
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Функция изменения матрицы
__global__ void iterate(double* A, double* A_new, size_t size_x, size_t size_y) {
	size_t j = blockIdx.x * blockDim.x + threadIdx.x;
	size_t i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((j > 0) && (i > 0) && (i < size_y - 1) && (j < size_x - 1))  // Don't update borders
		A_new[i * size_x + j] = 0.25 * (A[i * size_x + j - 1] + A[(i - 1) * size_x + j] + A[(i + 1) * size_x + j] + A[i * size_x + j + 1]);	
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Функция разницы матриц
__global__ void subtraction(double* A, double* A_new, double* A_err, size_t size_x) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if ((j > 0) && (i > 0) && (i < size_x - 1) && (j < size_x - 1))
		A_err[i * size_x + j] = A[i * size_x + j] - A_new[i * size_x + j];
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Значения по умодчанию
double eps = 1E-6;
int size = 256;
int iter_max = 1E6;

int main(int argc, char** argv) {
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Получение значений из командной строки
	sscanf(argv[1], "%lf", &eps);
	sscanf(argv[2], "%d", &size);
	sscanf(argv[3], "%d", &iter_max);


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Выбор видеокарт
	int DEVICE, COUNT_DEVICE;
	MPI_Init(&argc, &argv);

	MPI_Comm_rank(MPI_COMM_WORLD, &DEVICE);
	MPI_Comm_size(MPI_COMM_WORLD, &COUNT_DEVICE);

	hipSetDevice(DEVICE);

	std::cout << "Number of processes: " << COUNT_DEVICE << std::endl;
	if (DEVICE == 0)
		std::cout << "Settings: " << "\n\tMin error: " << eps << "\n\tMax iteration: " << iter_max << "\n\tSize: " << size << "x" << size << std::endl;

	size_t size_y; 
	
	size_y = find_area(size,COUNT_DEVICE,DEVICE);
	 size_t start_idx = size_y * DEVICE;
    if (DEVICE == COUNT_DEVICE - 1) {
        start_idx = (size - size_y) ;
    }
	//size_y = size / COUNT_DEVICE;
	//if (COUNT_DEVICE > 1)
	//	size_y++;
	int prc_area_add = 0;
    if (COUNT_DEVICE > 1)
    {
        if (DEVICE != 0 && DEVICE != COUNT_DEVICE -1){
            prc_area_add = 2;
        }
        else{
            prc_area_add++;
        }
    }
    size_y = size_y + prc_area_add;
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Выделения памяти
	double *A, *A_Device, *A_new_Device, *A_error_Device, *deviceError, *tempStorage = NULL;
	size_t tempStorageSize = 0;

	hipHostMalloc(&A, sizeof(double) * size * size);
	for (int i = 0; i < size; i++)  {
		A[i] = 10.0 + i * 10.0 / (size - 1);
		A[i * size] = 10.0 + i * 10.0 / (size - 1);
		A[size - 1 + i * size] = 20.0 + i * 10.0 / (size - 1);
		A[size * (size - 1) + i] = 20.0 + i * 10.0 / (size - 1);
	}

	dim3 threads(THREAD,1);
	dim3 blocks(size/THREAD, size_y);

	hipMalloc(&A_Device, sizeof(double) * size * size_y);
	hipMalloc(&A_new_Device, sizeof(double) * size * size_y);
	hipMalloc(&A_error_Device, sizeof(double) * size * size_y);
	hipMalloc(&deviceError, sizeof(double));

	size_t offset = (DEVICE != 0) ? size : 0;
 	hipMemcpy(A_Device, A + (start_idx*size) - offset, sizeof(double) * size * size_y, hipMemcpyHostToDevice);
	hipMemcpy(A_new_Device, A + (size * start_idx) - offset, sizeof(double) * size * size_y, hipMemcpyHostToDevice);

	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, A_error_Device, deviceError, size * size_y);
	hipMalloc(&tempStorage, tempStorageSize);
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Основной цикл
	size_t iter = 0;
	double error = 1.0;
	clock_t begin = clock();
	hipStream_t stream;
	hipStreamCreate(&stream);

	while((iter < iter_max) && error > eps)	{
		iterate<<<blocks, threads, 0, stream>>>(A_Device, A_new_Device, size, size_y);
		iter++;
		// Расчитываем ошибку каждую итерацию кратную размеру матрицы
		if (iter % UPDATE == 0 ) {
			
			subtraction<<<blocks, threads, 0, stream>>>(A_new_Device, A_Device, A_error_Device, size);
			hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, A_error_Device, deviceError, size * size_y, stream);
			hipStreamSynchronize(stream);

			if (COUNT_DEVICE > 1) {
			hipMemcpyAsync(&error, deviceError, sizeof(double), hipMemcpyDeviceToHost, stream);
				//передаём ошибку всем процессам
				MPI_Allreduce((void*)&error, (void*)&error, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
			}
			else
				hipMemcpy(&error, deviceError, sizeof(double), hipMemcpyDeviceToHost);

		}

		if (DEVICE > 0)			   // Обмен верхней границей
			MPI_Sendrecv(A_new_Device + size + 1, size - 2, MPI_DOUBLE, DEVICE - 1, 0, A_new_Device + 1, size - 2, MPI_DOUBLE, DEVICE - 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);

		if (DEVICE < COUNT_DEVICE - 1) // Обмен нижней границей
			MPI_Sendrecv(A_new_Device + (size_y - 2) * size + 1, size - 2, MPI_DOUBLE, DEVICE + 1, 0, A_new_Device + (size_y - 1) * size + 1, size - 2, MPI_DOUBLE, DEVICE + 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);


		std::swap(A_Device, A_new_Device);
	}

	clock_t end = clock();
	if (DEVICE == 0) {
		std::cout << "Result:\n\tIter: " << iter << "\n\tError: " << error << "\n\tTime: " << 1.0 * (end - begin) / CLOCKS_PER_SEC << std::endl;
	}
/////////////////////////////////////

hipMemcpy(A, A_Device, sizeof(double) * size * size_y, hipMemcpyDeviceToHost);
    MPI_Barrier(MPI_COMM_WORLD);
    if (DEVICE == 0)
    {std::cout << "DEVICE: " << DEVICE << std::endl << std::endl;
    {    for (int i = 0; i < size_y - prc_area_add; i ++) {
            for (int j = 0; j < size; j ++) {
                std::cout << A[i * size + j] << " ";
            }
            std::cout << std::endl;
        }
        // std::cout << std::endl;
    }}

    if (DEVICE == 1)
    {std::cout << "DEVICE: " << DEVICE << std::endl << std::endl;
    {    for (int i = 1; i < size_y - prc_area_add + 1; i ++) {
            for (int j = 0; j < size; j ++) {
                std::cout << A[i * size + j] << " ";
            }
            std::cout << std::endl;
        }
        // std::cout << std::endl;
    }}
    if (DEVICE == 2)
    {std::cout << "DEVICE: " << DEVICE << std::endl << std::endl;
    {    for (int i = 1; i < size_y - prc_area_add + 1; i ++) {
            for (int j = 0; j < size; j ++) {
                std::cout << A[i * size + j] << " ";
            }
            std::cout << std::endl;
        }
        // std::cout << std::endl;
    }}
    if (DEVICE == 3)
    {std::cout << "DEVICE: " << DEVICE << std::endl << std::endl;
    {    for (int i = 1; i < size_y; i ++) {
            for (int j = 0; j < size; j ++) {
                std::cout << A[i * size + j] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }}



//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Чистка памяти
	hipFree(A_Device);
	hipFree(A_new_Device);
	hipFree(A_error_Device);
	hipFree(tempStorage);
	hipStreamDestroy(stream);
	MPI_Finalize();

	return 0;
}
