#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <sstream>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

void complet(double* A,int size){
	double step = 10 / ((double)size - 1);
	for (int i = 0; i < size; i++)
	{
		A[i] = 10.0 + i * step;
		A[i * size] = 10.0 + i * step;
		A[size - 1 + i * size] = 20.0 + i * step;
		A[size * (size - 1) + i] = 20.0 + i * step;
	}
}

void printArr(double* A,int size){
	for (int i = 0; i < size; i++){
		for (int j = 0; j < size; j++){
		std::cout<< A[i * size+j]<<" ";
		}
		std::cout<<std::endl;
	}
}

__global__ void step(double* A, double* A_new, size_t size) {
	size_t i = blockIdx.x + 1, j = threadIdx.x + 1;
	A_new[i * size + j] = 0.25 * (A[i * size + j - 1] + A[(i - 1) * size + j] /
	 + A[(i + 1) * size + j] + A[i * size + j + 1]);	
	}

__global__ void subtraction(double* A, double* A_new, double* A_err) {
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	A_err[idx] = A[idx] - A_new[idx];
}

int main(int argc, char** argv){
	std::cout<< "Cuda"<<std::endl;
	clock_t start = clock();
    int itermax;
    int size;
	double tol;

	if (argc != 4 || atof(argv[1])>1  || atof(argv[3])<1)
    {
        printf("Недостаточно параметров. Нужно ввести 3 параметра\n");
        printf("1.точность;\n2. размер сетки;\n3. количество итераций\n");
        return 0;
    }

	tol = atof(argv[1]);
	size = atof(argv[2]);
	itermax = atof(argv[3]);
	size_t totalSize = size * size;

	double* A= NULL;
    double* A_D, *A_D_new, *A_D_err, *deviceError, *tempStorage = NULL;
	size_t tempStorageSize = 0;

	A = (double*)calloc(totalSize, sizeof(double));

	hipMalloc(&A_D, sizeof(double) * totalSize);
	hipMalloc(&A_D_new, sizeof(double) * totalSize);
	hipMalloc(&A_D_err, sizeof(double) * totalSize);
	hipMalloc(&deviceError, sizeof(double) );

	hipMemcpy(A_D_err, A, sizeof(double) * totalSize, hipMemcpyHostToDevice);

	complet(A,size);

	hipMemcpy(A_D, A, sizeof(double) * totalSize, hipMemcpyHostToDevice);
	hipMemcpy(A_D_new, A, sizeof(double) * totalSize, hipMemcpyHostToDevice);

	hipStream_t stream;
	hipStreamCreate(&stream);
	hipGraph_t graph;
	hipGraphExec_t graph_instance;

	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, A_D_err, deviceError, totalSize, stream);
	hipMalloc(&tempStorage, tempStorageSize);
///////////////////////////////////////////////////////////////////// Создание графа
	hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

	for (size_t i = 0; i < size; i += 2) {
		step<<<size - 2, size - 2, 0, stream>>>(A_D, A_D_new, size);
		//split(A_D, A_D_new);
		step<<<size - 2, size - 2, 0, stream>>>(A_D_new, A_D, size);
	}

	subtraction<<<size, size, 0, stream>>>(A_D, A_D_new,A_D_err);
	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, A_D_err, deviceError, totalSize, stream);

	hipStreamEndCapture(stream, &graph);
	hipGraphInstantiate(&graph_instance, graph, NULL, NULL, 0);
///////////////////////////////////////////////////////////////////// Основной цикл
	int iter = 0; 
	double error = 1 ;
	 do{
		hipGraphLaunch(graph_instance, stream);
		hipMemcpy(&error, deviceError, sizeof(double), hipMemcpyDeviceToHost);
		//hipMemcpyFromSymbol(&error, HIP_SYMBOL(deviceError), sizeof(double), hipMemcpyDeviceToHost);
		iter += size;
		//std::cout <<error<< std::endl;
	}while(iter < itermax && error > tol);
	std::cout << "Result:\n\tIter: " << iter << "\n\tError: " << error << std::endl;

	//printArr(A,size);

	free(A);
	hipFree(A_D);
	hipFree(A_D_new);
	hipFree(A_D_err);
	hipFree(deviceError);
	hipFree(tempStorage);
	hipGraphDestroy(graph);
	hipStreamDestroy(stream);
	
	clock_t end = clock();
    double seconds = (double)(end - start) / CLOCKS_PER_SEC;
    printf("The time: %f seconds\n", seconds);
	return 0;
}